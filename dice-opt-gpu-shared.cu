
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <ctype.h>
#include <errno.h>

#define THREADS_ALIGN 32

/* strが有効なuint32_t型の範囲の正の整数かを返す */
int is_valid_pnum(const char* str) {
	char* end;
	long res;
	if (str == NULL) return 0;
	errno = 0;
	res = strtol(str, &end, 10);
	return *end == '\0' && errno == 0 && 1 <= res && (unsigned long)res <= UINT32_MAX;
}

/* オーバーフローに気をつけながら、size_tにuint32_tを掛ける */
size_t multiply_size(size_t size, uint32_t m) {
	if (size > SIZE_MAX / m) {
		fprintf(stderr, "size to allocate too big!\n");
		exit(1);
	}
	return size * m;
}

/* dst += src; 最後の繰り上がりの数を返す */
uint32_t add_nums(uint32_t* dst, const uint32_t* src, uint32_t length) {
	uint32_t carry = 0;
	uint32_t i;
	for (i = 0; i < length; i++) {
		uint32_t next_carry = (dst[i] > UINT32_MAX - src[i]);
		dst[i] += src[i];
		next_carry += (dst[i] > UINT32_MAX - carry);
		dst[i] += carry;
		carry = next_carry;
	}
	return carry;
}
__inline__ __device__ uint32_t add_nums_gpu(uint32_t* dst, const uint32_t* src, uint32_t length) {
	uint32_t carry = 0;
	uint32_t i;
	for (i = 0; i < length; i++) {
		uint32_t next_carry = (dst[i] > UINT32_MAX - src[i]);
		dst[i] += src[i];
		next_carry += (dst[i] > UINT32_MAX - carry);
		dst[i] += carry;
		carry = next_carry;
	}
	return carry;
}

/* out = in * mul; 最後の繰り上がりの数を返す */
uint32_t multiply(uint32_t* out, const uint32_t* in, uint32_t mul, uint32_t length) {
	uint32_t i;
	uint32_t carry = 0;
	for (i = 0; i < length; i++) {
		uint64_t next = (uint64_t)in[i] * mul + carry;
		out[i] = (uint32_t)(next & UINT32_C(0xffffffff));
		carry = (uint32_t)(next >> 32);
	}
	return carry;
}

/* 指定されたサイコロのパターン数を表すのに必要十分なDWORD数を返す */
/* 返す数はalignmentの倍数とする */
uint32_t get_required_dwords(
uint32_t dice_num, uint32_t dice_max_output, uint32_t alignment) {
	uint32_t* calc_buffer = (uint32_t*)malloc(sizeof(uint32_t));
	uint32_t calc_buffer_len = 1;
	uint32_t i;
	uint32_t carry;
	uint64_t ret;
	if (calc_buffer == NULL) {
		perror("get_required_bytes : malloc");
		exit(1);
	}
	calc_buffer[0] = 1;
	for (i = 0; i < dice_num; i++) {
		carry = multiply(calc_buffer, calc_buffer, dice_max_output, calc_buffer_len);
		if (carry > 0) {
			uint32_t* next_buffer = (uint32_t*)realloc(calc_buffer,
				sizeof(uint32_t) * ++calc_buffer_len);
			if (next_buffer == NULL) {
				perror("get_required_bytes : realloc");
				free(calc_buffer);
				exit(1);
			}
			calc_buffer = next_buffer;
			calc_buffer[calc_buffer_len - 1] = carry;
		}
	}
	free(calc_buffer);

	ret = (((uint64_t)calc_buffer_len + alignment - 1) / alignment)
		* alignment;
	if (ret > UINT32_MAX) {
		fprintf(stderr, "get_required_bytes : size too big\n");
		exit(1);
	}
	return (uint32_t)ret;
}

struct result_data {
	uint32_t dice_num, dice_max_output;
	uint32_t *puttern_count, *all_puttern_count;
};

uint32_t cmp_size, cmp_size2;
uint32_t *cmp_buffer, *cmp_buffer_a, *cmp_buffer_b, *cmp_buffer_temp;
int result_data_cmp(const void* x, const void* y) {
	const struct result_data *a = (const struct result_data*)x;
	const struct result_data *b = (const struct result_data*)y;
	uint32_t i;
	/* cmp_buffer_a = a->puttern_count * b->all_puttern_count */
	cmp_buffer_a[cmp_size] = multiply(cmp_buffer_a, a->puttern_count,
		b->all_puttern_count[0], cmp_size);
	for (i = 1; i < cmp_size; i++) {
		cmp_buffer_a[(size_t)cmp_size + i] = multiply(cmp_buffer_temp,
			a->puttern_count, b->all_puttern_count[i], cmp_size);
		cmp_buffer_a[(size_t)cmp_size + i] += add_nums(cmp_buffer_a + i,
			cmp_buffer_temp, cmp_size);
	}
	/* cmp_buffer_b = b->puttern_count * a->all_puttern_count */
	cmp_buffer_b[cmp_size] = multiply(cmp_buffer_b, b->puttern_count,
		a->all_puttern_count[0], cmp_size);
	for (i = 1; i < cmp_size; i++) {
		cmp_buffer_b[(size_t)cmp_size + i] = multiply(cmp_buffer_temp,
			b->puttern_count, a->all_puttern_count[i], cmp_size);
		cmp_buffer_b[(size_t)cmp_size + i] += add_nums(cmp_buffer_b + i,
			cmp_buffer_temp, cmp_size);
	}\
	/* 掛け算の結果の比較 (降順) */
	for (i = 0; i < cmp_size2; i++) {
		if (cmp_buffer_a[cmp_size2 - i - 1] > cmp_buffer_b[cmp_size2 - i - 1])
			return -1;
		if (cmp_buffer_a[cmp_size2 - i - 1] < cmp_buffer_b[cmp_size2 - i - 1])
			return 1;
	}
	/* サイコロを振る数の比較 (昇順) */
	if (a->dice_num > b->dice_num) return 1;
	if (a->dice_num < b->dice_num) return -1;
	/* サイコロで出る目の最大値の比較 (昇順) */
	if (a->dice_max_output > b->dice_max_output) return 1;
	if (a->dice_max_output< b->dice_max_output) return -1;
	return 0;
}

/* 多倍長整数をdouble型に変換する */
double num_to_double(const uint32_t* num, uint32_t length) {
	double res = 0;
	uint32_t i;
	for (i = 0; i < length; i++) {
		res = res * (UINT32_MAX + 1.0) + num[length - i - 1];
	}
	return res;
}

/* 多倍長整数を10進数で出力する。入力の多倍長整数は破壊される。 */
void print_num_and_destroy(uint32_t* num, uint32_t length) {
	static const uint32_t divisor = UINT32_C(1000000000);
	uint32_t remainder = 0;
	uint32_t i;
	int nonzero_exists = 0;
	for (i = 0; i < length; i++) {
		uint64_t current = ((uint64_t)remainder << 32) + num[length - i - 1];
		if ((num[length - i - 1] = (uint32_t)(current /divisor)) != 0) nonzero_exists = 1;
		remainder = (uint32_t)(current % divisor);
	}
	if (nonzero_exists) {
		print_num_and_destroy(num, length);
		printf("%09"PRIu32, remainder);
	} else {
		printf("%"PRIu32, remainder);
	}
}

/* 初期化 */
__global__ void init_dp(uint32_t* calculate_src, uint32_t required_dwords) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	for (uint32_t j = 0; j < required_dwords; j++) {
		calculate_src[i * (size_t)required_dwords + j] = 0;
	}
	/* サイコロを0個振った時、合計は0の1通りのみ */
	if (i == 0) {
		calculate_src[0] = 1;
	}
}

__global__ void run_dp(uint32_t* calculate_src, uint32_t* calculate_dst,
uint32_t required_dwords, uint32_t dice_max) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	/* その数について、パターン数を計算する */
	extern __shared__ uint32_t dst_buffer[];
	/* 足し算のために初期化する */
	for (uint32_t j = 0; j < required_dwords; j++) {
		dst_buffer[threadIdx.x * (size_t)required_dwords + j] = 0;
	}
	/* パターン数の計算のため、前のパターン数を足す */
	for (uint32_t j = 0; j <= dice_max; j++) {
		if (i > j) {
			add_nums_gpu(
				&dst_buffer[threadIdx.x * (size_t)required_dwords],
				&calculate_src[(i - j - 1) * (size_t)required_dwords],
				required_dwords);
		}
	}
	/* 結果を書き込む */
	for (uint32_t j = 0; j < required_dwords; j++) {
		calculate_dst[i * (size_t)required_dwords + j] =
			dst_buffer[threadIdx.x * (size_t)required_dwords + j];
	}
}

int main(int argc, char* argv[]) {
	/* 入力パラメータ */
	uint32_t dice_max_num, dice_max_output, target_value, output_num;
	/* パターン数を表すのに使うDWORD数 */
	uint32_t required_dwords;
	/* 計算バッファ */
	uint32_t *calculate_buffer, *calculate_src, *calculate_dst, *tmp;
	size_t calculate_buffer_size;
	size_t calculate_buffer_offset;
	/* 計算に用いる変数 */
	uint32_t dice_max_sum;
	uint32_t i, j, k;
	/* 結果の保存に用いる変数 */
	struct result_data *results, *current_result;
	uint32_t *result_putterns, *result_all_putterns;
	uint32_t result_count = 0;
	uint32_t *previous_all_putterns = NULL;

	/* 入力を読み取る */
	if ((argc != 4 && argc != 5) || !is_valid_pnum(argv[1]) ||
	!is_valid_pnum(argv[2]) || !is_valid_pnum(argv[3]) ||
	(argc >= 5 && !is_valid_pnum(argv[4]))) {
		fprintf(stderr,
			"Usage: %s max-num-of-dice max-output-of-a-die target-value [output-num]\n",
			argc >= 1 ? argv[0] : "dice-opt");
		return 1;
	}
	dice_max_num = (uint32_t)atol(argv[1]);
	dice_max_output = (uint32_t)atol(argv[2]);
	target_value = (uint32_t)atol(argv[3]);
	output_num = argc >= 5 ? (uint32_t)atol(argv[4]) : UINT32_C(10);
	printf("calculating putterns that becomes %"PRIu32"\n"
		"with max %"PRIu32" dice whose output are upto max %"PRIu32"\n",
		target_value, dice_max_num, dice_max_output);

	/* 計算バッファを確保する */
	if (dice_max_num > UINT32_MAX / dice_max_output) {
		fputs("parameter too big!\n", stderr);
		return 1;
	}
	dice_max_sum = dice_max_output * dice_max_num;
	if (dice_max_sum >= UINT32_MAX - ((THREADS_ALIGN - dice_max_sum % THREADS_ALIGN) % THREADS_ALIGN)) {
		fputs("parameter too big!\n", stderr);
		return 1;
	}
	if (target_value > dice_max_sum) {
		puts("there is no chance because target is too big");
		return 0;
	}
	dice_max_sum++;
	if (dice_max_sum % THREADS_ALIGN != 0) {
		uint32_t delta = THREADS_ALIGN - dice_max_sum % THREADS_ALIGN;
		if (dice_max_sum > UINT32_MAX - delta) {
			fputs("parameter too big!\n", stderr);
			return 1;
		}
		dice_max_sum += delta;
	}
	required_dwords = get_required_dwords(dice_max_num, dice_max_output, 1);
	calculate_buffer_offset = multiply_size(
			multiply_size(sizeof(uint32_t), required_dwords), dice_max_sum);
	calculate_buffer_size = multiply_size(calculate_buffer_offset, 2);
	hipMalloc(&calculate_buffer, calculate_buffer_size);
	calculate_src = calculate_buffer;
	calculate_dst = calculate_buffer +
		(calculate_buffer_offset / sizeof(uint32_t));

	/* 結果の保存に用いるバッファの確保を行う */
	results = (struct result_data*)malloc(
		multiply_size(sizeof(struct result_data), dice_max_sum));
	if (results == NULL) {
		perror("failed to allocate result buffer");
		free(calculate_buffer);
		return 1;
	}
	result_putterns = (uint32_t*)malloc(
		multiply_size(
			multiply_size(sizeof(uint32_t), dice_max_sum), required_dwords));
	if (result_putterns == NULL) {
		perror("failed to allocate result puttern buffer");
		free(calculate_buffer);
		free(results);
		return 1;
	}
	result_all_putterns = (uint32_t*)malloc(
		multiply_size(
			multiply_size(sizeof(uint32_t), dice_max_sum), required_dwords));
	if (result_putterns == NULL) {
		perror("failed to allocate result all puttern buffer");
		free(calculate_buffer);
		free(results);
		free(result_putterns);
		return 1;
	}

	/* 比較用のメモリを確保する */
	cmp_size = required_dwords;
	if (cmp_size > UINT32_MAX / 2) {
		fputs("size too big!\n", stderr);
		free(calculate_buffer);
		free(results);
		free(result_putterns);
		free(result_all_putterns);
		return 1;
	}
	cmp_size2 = cmp_size * 2;
	cmp_buffer = (uint32_t*)malloc(
		multiply_size(
			multiply_size(sizeof(uint32_t), required_dwords), 5));
	if (cmp_buffer == NULL) {
		perror("failed to allocate compare buffer");
		free(calculate_buffer);
		free(results);
		free(result_putterns);
		free(result_all_putterns);
		return 1;
	}
	cmp_buffer_a = cmp_buffer;
	cmp_buffer_b = cmp_buffer + (size_t)required_dwords * 2;
	cmp_buffer_temp = cmp_buffer + (size_t)required_dwords * 4;

	/* 計算を行う */
	/* サイコロの目の最大値を全探索する */
	for (i = 0; i < dice_max_output; i++) {
		init_dp<<<dice_max_sum / THREADS_ALIGN, THREADS_ALIGN>>>
			(calculate_src, required_dwords);
		hipDeviceSynchronize();

		/* DP */
		run_dp<<<dice_max_sum / THREADS_ALIGN, THREADS_ALIGN,
			sizeof(uint32_t) * THREADS_ALIGN * required_dwords>>>
			(calculate_src, calculate_dst, required_dwords, i);
		/* サイコロの数繰り返す */
		for (j = 0; j < dice_max_num; j++) {
			/* パターン数を記録する */
			current_result = &results[result_count++];
			current_result->dice_num = j + 1;
			current_result->dice_max_output = i + 1;
			current_result->puttern_count =
				&result_putterns
					[(i * (size_t)dice_max_num + j) * required_dwords];
			current_result->all_puttern_count =
				&result_all_putterns
					[(i * (size_t)dice_max_num + j) * required_dwords];
			if (j == 0) {
				current_result->all_puttern_count[0] = i + 1;
				for (k = 1; k < required_dwords; k++) {
					current_result->all_puttern_count[k] = 0;
				}
			} else {
				multiply(current_result->all_puttern_count,
					previous_all_putterns, i + 1, required_dwords);
			}
			previous_all_putterns = current_result->all_puttern_count;
			hipDeviceSynchronize();
			hipMemcpy(current_result->puttern_count,
				&calculate_dst[target_value * (size_t)required_dwords],
				sizeof(*current_result->puttern_count) * required_dwords,
				hipMemcpyDeviceToHost);
			if (j + 1 < dice_max_num) {
				/* バッファを入れ替える */
				tmp = calculate_src;
				calculate_src = calculate_dst;
				calculate_dst = tmp;
				/* 次の計算を行う */
				run_dp<<<dice_max_sum / THREADS_ALIGN, THREADS_ALIGN,
					sizeof(uint32_t) * THREADS_ALIGN * required_dwords>>>
					(calculate_src, calculate_dst, required_dwords, i);
			}
			/* バッファの挿入ソートを行う */
			for (k = result_count - 1; k > 0; k--) {
				if (result_data_cmp(&results[k - 1], &results[k]) > 0) {
					struct result_data tmp = results[k - 1];
					results[k - 1] = results[k];
					results[k] = tmp;
				} else {
					break;
				}
			}
			/* 表示範囲からあふれた無駄なデータを削る */
			if (result_count > output_num) result_count = output_num;
		}
	}
	hipFree(calculate_buffer);
	free(cmp_buffer);

	/* 上位を出力する */
	for (i = 0; i < dice_max_sum && i < result_count; i++) {
		printf("%"PRIu32"d%"PRIu32" -> %g (",
			results[i].dice_num, results[i].dice_max_output,
			num_to_double(results[i].puttern_count, required_dwords) /
				num_to_double(results[i].all_puttern_count, required_dwords));
		print_num_and_destroy(results[i].puttern_count, required_dwords);
		printf(" / ");
		print_num_and_destroy(results[i].all_puttern_count, required_dwords);
		printf(")\n");
	}

	free(results);
	free(result_putterns);
	free(result_all_putterns);
	return 0;
}
